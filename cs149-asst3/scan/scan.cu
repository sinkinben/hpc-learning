#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"
#include "debug.h"

#define THREADS_PER_BLOCK 256

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void upSweep(int *arr, int N, int d)
{
    int d2 = d * 2;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = tid * d2;
    if (idx < N)
        arr[idx + d2 - 1] += arr[idx + d - 1];
}

__global__ void downSweep(int *arr, int N, int d)
{
    int d2 = d * 2;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = tid * d2;
    if (idx < N)
    {
        int t = arr[idx + d - 1];
        arr[idx + d - 1] = arr[idx + d2 - 1];
        arr[idx + d2 - 1] += t;
    }
}

// exclusive_scan --
void exclusive_scan(int *input, int N, int *result)
{
    constexpr int blockDim = 512;

    // up sweep
    for (int d = 1; d <= N / 2; d *= 2)
    {
        int gridDim = (N / (d * 2)) / blockDim + 1;
        upSweep<<<gridDim, blockDim>>>(result, N, d);
    }
    cudaCheckError(hipDeviceSynchronize());

    int zero = 0;
    hipMemcpy(&result[N - 1], &zero, sizeof(int), hipMemcpyHostToDevice);

    // down sweep
    for (int d = N / 2; d >= 1; d /= 2)
    {
        int gridDim = (N / (d * 2)) / blockDim + 1;
        downSweep<<<gridDim, blockDim>>>(result, N, d);
    }
}

//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int *inarray, int *end, int *resultarray)
{
    int *device_result;
    int *device_input;
    int N = end - inarray;

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);

    cudaCheckError(hipMalloc((void **)&device_result, sizeof(int) * rounded_length));
    cudaCheckError(hipMalloc((void **)&device_input, sizeof(int) * rounded_length));
    assert(device_input != nullptr && device_result != nullptr);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, rounded_length, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    hipFree(device_input);
    hipFree(device_result);
    return overallDuration;
}

// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int *inarray, int *end, int *resultarray)
{

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration;
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int *device_input, int length, int *device_output)
{

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    return 0;
}

//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length)
{

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);

    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();

    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime;
    return duration;
}

void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
